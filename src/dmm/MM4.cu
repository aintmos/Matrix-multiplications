#include "hip/hip_runtime.h"
#include "GPUdebug.hpp"
#include "matConst.hpp"

using namespace std;

__global__ void MM_Kernel_Kernel(dataType* matrix, dataType* input, dataType* res){
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if(i >= rowNumC || j >= colNumC) return;

    int acc = 0;
    for(int k = 0; k < colNumA; ++k){
        acc += matrix[i * colNumA + k] * input[k * colNumB + j];
    }
    res[i * colNumC + j] = acc;
}

void MM_Kernel(dataType** matrix, dataType** input, dataType** res){
    dataType *matrix_GPU;
    dataType *input_GPU;
    dataType *res_GPU;
    HANDLE_ERROR(hipMalloc(&matrix_GPU, sizeof(dataType)*rowNumA*colNumA));
    HANDLE_ERROR(hipMalloc(&input_GPU,  sizeof(int)*rowNumB*colNumB));
    HANDLE_ERROR(hipMalloc(&res_GPU,    sizeof(int)*rowNumC*colNumC));
    for(int i = 0; i < rowNumA; ++i){
        HANDLE_ERROR(hipMemcpy(matrix_GPU + colNumA * i, matrix[i], sizeof(dataType)*colNumA, hipMemcpyHostToDevice));
    }
    for(int i = 0; i < rowNumB; ++i){
        HANDLE_ERROR(hipMemcpy(input_GPU  + colNumB * i, input[i],  sizeof(int)*colNumB, hipMemcpyHostToDevice));
    }

    dim3 threadDim(rowNumC > subBlockSize ? subBlockSize : rowNumC , colNumC > subBlockSize ? subBlockSize : colNumC);
    dim3 blockDim((rowNumC + subBlockSize - 1)/subBlockSize , (colNumC + subBlockSize - 1)/subBlockSize);
    MM_Kernel_Kernel<<<blockDim, threadDim>>>(matrix_GPU, input_GPU, res_GPU);

    for(int i = 0; i < rowNumC; ++i){
        HANDLE_ERROR(hipMemcpy(res[i], res_GPU + colNumC * i, sizeof(dataType)*colNumC, hipMemcpyDeviceToHost));
    }
    HANDLE_ERROR(hipFree(matrix_GPU));
    HANDLE_ERROR(hipFree(input_GPU));
    HANDLE_ERROR(hipFree(res_GPU));
}
