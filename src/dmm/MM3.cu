#include "hip/hip_runtime.h"
#include "GPUdebug.hpp"
#include "matConst.hpp"

using namespace std;

__global__ void MM_Kernel_Kernel(dataType* matrix, dataType* input, dataType* res){
    int global = threadIdx.x + blockIdx.x * blockDim.x;
    if(global > rowNumC * colNumC) return;
    int i = global/colNumC;
    int j = global%colNumC;
    int acc = 0;
    for(int k = 0; k < colNumA; ++k){
        acc += matrix[i * colNumA + k] * input[k * colNumB + j];
    }
    res[global] = acc;
}

void MM_Kernel(dataType** matrix, dataType** input, dataType** res){
    dataType *matrix_GPU;
    dataType *input_GPU;
    dataType *res_GPU;
    HANDLE_ERROR(hipMalloc(&matrix_GPU, sizeof(dataType)*rowNumA*colNumA));
    HANDLE_ERROR(hipMalloc(&input_GPU,  sizeof(int)*rowNumB*colNumB));
    HANDLE_ERROR(hipMalloc(&res_GPU,    sizeof(int)*rowNumC*colNumC));
    for(int i = 0; i < rowNumA; ++i){
        HANDLE_ERROR(hipMemcpy(matrix_GPU + colNumA * i, matrix[i], sizeof(dataType)*colNumA, hipMemcpyHostToDevice));
    }
    for(int i = 0; i < rowNumB; ++i){
        HANDLE_ERROR(hipMemcpy(input_GPU + colNumB * i, input[i], sizeof(dataType)*colNumB, hipMemcpyHostToDevice));
    }

    int work = rowNumC * colNumC;
    MM_Kernel_Kernel<<<(work + 1023)/1024, work > 1024? 1024 : work >>>(matrix_GPU, input_GPU, res_GPU);

    for(int i = 0; i < rowNumC; ++i){
        HANDLE_ERROR(hipMemcpy(res[i], res_GPU + colNumC * i, sizeof(dataType)*colNumC, hipMemcpyDeviceToHost));
    }
    HANDLE_ERROR(hipFree(matrix_GPU));
    HANDLE_ERROR(hipFree(input_GPU));
    HANDLE_ERROR(hipFree(res_GPU));
}
