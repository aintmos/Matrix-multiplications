#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <set>
#include "GPUdebug.hpp"
#include "papi.h"

using namespace std;


//A(Sparse) B = C
constexpr int rowNumA = 2048;
constexpr int colNumA = 1024;
constexpr int rowNumB = colNumA;
constexpr int colNumB = 2048;
constexpr int rowNumC = rowNumA;
constexpr int colNumC = colNumB;
constexpr int maxEdge = 100;

void SPMM_CPU(int* rowPtr, int* colIdx, int* value, int** input, int** res){
    for(int i = 0; i < rowNumA; ++i){
        for(int j = rowPtr[i]; j < rowPtr[i + 1]; ++j){
            for(int k = 0; k < colNumB; ++k){
                res[i][k] += value[j] * input[colIdx[j]][k];
            }
        }
    }
}

__global__ void SPMM_Kernel_kernel(int* rowPtr, int* colIdx, int* value, int** input, int** res){
    for(int i = 0; i < rowNumA; ++i){
        for(int j = rowPtr[i]; j < rowPtr[i + 1]; ++j){
            for(int k = 0; k < colNumB; ++k){
                res[i][k] += value[j] * input[colIdx[j]][k];
            }
        }
    }
}

void SPMM_Kernel(int* rowPtr, int* colIdx, int* value, int** input, int** res){
    int* rowPtr_GPU;
    int* colIdx_GPU;
    int* value_GPU;
    int* input_GPU;
    int* res_GPU;
    
    int numNZ = rowPtr[rowNumA];

    HANDLE_ERROR(hipMalloc(&rowPtr_GPU, sizeof(dataType) * (rowNumA + 1)));
    HANDLE_ERROR(hipMalloc(&colIdx_GPU, sizeof(dataType) * numNZ));
    HANDLE_ERROR(hipMalloc(&value_GPU, sizeof(dataType) * numNZ));
    HANDLE_ERROR(hipMalloc(&input_GPU, sizeof(dataType) * rowNumA * rowNumA));
    HANDLE_ERROR(hipMalloc(&res_GPU, sizeof(dataType) * rowNumA * rowNumA));

    HANDLE_ERROR(hipMemcpy(rowPtr_GPU, rowPtr, sizeof(dataType) * (rowNumA + 1), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(colIdx_GPU, colIdx, sizeof(dataType) * numNZ, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(value_GPU, value, sizeof(dataType) * numNZ, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(input_GPU, input, sizeof(dataType) * rowNumA * rowNumA, hipMemcpyDeviceToHost));

    
    
    HANDLE_ERROR(hipMemcpy(res, res_GPU, sizeof(dataType) * rowNumA * rowNumA, hipMemcpyHostToDevice));
    
    HANDLE_ERROR(hipFree(rowPtr_GPU));
    HANDLE_ERROR(hipFree(colIdx_GPU));
    HANDLE_ERROR(hipFree(value_GPU));
      HANDLE_ERROR(hipFree(input_GPU));
   HANDLE_ERROR(hipFree(res_GPU));
}


int randomF(){
    return (rand()%100)/10.0f;
}

int main(int argc, char **argv){
    srand(time(NULL));
    int pre = 0;
    int* rowPtr = new int[rowNumA + 1];
    {
        for(int i = 0; i < rowNumA; ++i){
            int range = rand()%(maxEdge - 1) + 1;
            rowPtr[i] = pre;
            pre += range;
        }
        rowPtr[rowNumA] = pre;
    }
    int dataNum = pre;
    int* colIdx = new int[dataNum];
    int* value = new int[dataNum];
    
    for(int i = 0; i < rowNumA; ++i){
        int colNum = rowPtr[i + 1] - rowPtr[i];
        std::set<int> colIdxSet;
        for(int j = 0; j < colNum; ++j){
            int newColIdx;
            do{
                newColIdx = rand() % colNumA;
            }while(colIdxSet.find(newColIdx) != colIdxSet.end());
            colIdxSet.insert(newColIdx);
        }
        for(auto iter = std::make_pair(rowPtr[i], colIdxSet.begin()); iter.second != colIdxSet.end(); ++iter.first, ++iter.second){
            colIdx[iter.first] = *(iter.second);
            value[iter.first] = randomF();
        }
    }

    
    int **input = new int*[rowNumB];
    for(int i = 0; i < rowNumB; ++i){
        input[i] = new int[colNumB];
        for(int j = 0; j < colNumB; ++j){
            input[i][j] = randomF();
        }
    }
    
    int **res = new int*[rowNumC];
    for(int i = 0; i < rowNumC; ++i){
        res[i] = new int[colNumC];
        for(int j = 0; j < colNumC; ++j){
            res[i][j] = 0;
        }
    }


    SPMM_CPU(rowPtr, colIdx, value, input, res);

    for(int i = 0; i < rowNumB; ++i){
        delete[] input[i];
    }

    delete[] input;

    delete[] rowPtr;
    delete[] colIdx;
    delete[] value;

    for(int i = 0; i < rowNumC; ++i){
        delete[] res[i];
    }
    delete[] res;
    
    return 0;
}
